#include<hip/hip_runtime.h>
#include<stdio.h>
// Kernel definition
 __global__  void    MatAdd(float *A, float *B, float *C)
 { 
 int i = threadIdx.x; 
 int j = threadIdx.y; 
 *C= *A + *B;
 }
 int main() { 
 // Kernel invocation with one block of N * N * 1 threads
 int numBlocks = 1; 
 int i=0,j=0;
 float A=5,B=10,C;
 dim3 threadsPerBlock(10, 10);
 MatAdd<<<numBlocks, threadsPerBlock>>>(&A,&B,&C);
 printf("%f",C);
 return 1;
 }